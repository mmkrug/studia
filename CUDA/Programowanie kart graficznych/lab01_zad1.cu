#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""
#include <ctime>
#include <math.h>


#define  N   		10000000
#define  BLOCK_SIZE	1024

float 	   hArray[N];
float     *dArray;
int 	   blocks;


clock_t start_cpu, end_cpu;
double total_cpu;

clock_t start_gpu, end_gpu;
clock_t start_gpu_prolog, end_gpu_prolog;
clock_t start_gpu_epilog, end_gpu_epilog;
clock_t start_gpu_operacje, end_gpu_operacje;
double total_gpu;
double total_gpu_prolog;
double total_gpu_epilog;
double total_gpu_operacje;

void prologue(void) {
	hipMemset(hArray, 0, sizeof(hArray));
	for (int i = 0; i < N; i++) {
		hArray[i] = i + 1;
	}
	hipMalloc((void**)&dArray, sizeof(hArray));
	hipMemcpy(dArray, hArray, sizeof(hArray), hipMemcpyHostToDevice);
}

void epilogue(void) {
	hipMemcpy(hArray, dArray, sizeof(hArray), hipMemcpyDeviceToHost);
	hipFree(dArray);
}




// Kernel
__global__ void pow3(float *A) {
	int x = blockDim.x * blockIdx.x + threadIdx.x;

	if (x < N)
		A[x] = A[x] * A[x] * A[x] + A[x] * A[x] + A[x];
}


void pow3cpu(float *A) {

	for (int x = 0; x < N; x++) {
		A[x] = A[x] * A[x] * A[x] + A[x] * A[x] + A[x];
	}

}





int main(int argc, char** argv)
{
	int	 devCnt;

	hipGetDeviceCount(&devCnt);
	if (devCnt == 0) {
		perror("No CUDA devices available -- exiting.");
		return 1;
	}



	// CPU benchmark
	
	start_cpu = clock();
	{
		// CPU operacje
		for (int i = 0; i < N; i++) {
			hArray[i] = i + 1;
		}

		pow3cpu(hArray);
	}
	end_cpu = clock();
	total_cpu = (double)1000 * (end_cpu - start_cpu) / CLOCKS_PER_SEC;
	// koniec CPU



	// GPU benchmark
	start_gpu = clock();
	{
		// prolog
		start_gpu_prolog = clock();
		{
			prologue();
		}
		end_gpu_prolog = clock();
		// operacje na gpu
		start_gpu_operacje = clock();
		{
			blocks = N / BLOCK_SIZE;
			if (N % BLOCK_SIZE)
				blocks++;
			pow3 << <blocks, BLOCK_SIZE >> > (dArray);
			hipDeviceSynchronize();
		}
		end_gpu_operacje = clock();

		// epilog
		start_gpu_epilog = clock();
		{
			epilogue();
		}
		end_gpu_epilog = clock();

	}
	end_gpu = clock();


	total_gpu = (double)1000 * (end_gpu - start_gpu) / CLOCKS_PER_SEC;
	total_gpu_prolog = (double)1000 * (end_gpu_prolog - start_gpu_prolog) / CLOCKS_PER_SEC;
	total_gpu_operacje = (double)1000 * (end_gpu_operacje - start_gpu_operacje) / CLOCKS_PER_SEC;
	total_gpu_epilog = (double)1000 * (end_gpu_epilog - start_gpu_epilog) / CLOCKS_PER_SEC;


	printf("  Lab 1. Zad 1. Wektor\n\n");

	printf("  N:            %d \n", N);
	printf("  BLOCK SIZE:   %d \n", BLOCK_SIZE);
	printf("  Czas CPU:     %f [ms]\n", total_cpu);
	printf("  Czas GPU:     %f [ms]\n", total_gpu);
	printf("  GPU prolog:   %f [ms]\n", total_gpu_prolog);
	printf("  GPU operacje: %f [ms]\n", total_gpu_operacje);
	printf("  GPU epilog:   %f [ms]\n", total_gpu_epilog);



	return 0;
}
