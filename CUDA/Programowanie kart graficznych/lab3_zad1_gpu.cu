#include "hip/hip_runtime.h"
#include "EasyBMP.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
using namespace std;

#define	BLOCK_SIZE 32

int picWidth, picHeight;
BMP Input, Output;

struct Matrix {
	int width;
	int height;
	int *elementsRed;
	int *elementsGreen;
	int *elementsBlue;
};

__global__ void BmpToGrayKernel(const Matrix, Matrix);

void alokujMacierz(Matrix &M)
{
	M.width=picWidth;
	M.height=picHeight;
	M.elementsRed=(int*)malloc(M.width*M.height*sizeof(int));
	M.elementsGreen=(int*)malloc(M.width*M.height*sizeof(int));
	M.elementsBlue=(int*)malloc(M.width*M.height*sizeof(int));
}

void ConvertBmpToMatrix(BMP &Input, Matrix &M)
{
	for (int j=0; j<M.height; j++)
		for (int i=0; i<M.width; i++)
		{
			M.elementsRed[j*M.width+i]=Input(i,j)->Red;
			M.elementsGreen[j*M.width+i]=Input(i,j)->Green;
			M.elementsBlue[j*M.width+i]=Input(i,j)->Blue;
		}
}

void ConvertMatrixToBmp(BMP &Output, Matrix &M)
{
	for (int j=0; j<M.height; j++)
		for (int i=0; i<M.width; i++)
		{
			Output(i,j)->Red=(ebmpBYTE)M.elementsRed[j*M.width+i];
			Output(i,j)->Green=(ebmpBYTE)M.elementsGreen[j*M.width+i];
			Output(i,j)->Blue=(ebmpBYTE)M.elementsBlue[j*M.width+i];
		}
}

void zwolnijMacierz(Matrix &M)
{
	free(M.elementsRed);
	free(M.elementsGreen);
	free(M.elementsBlue);
}

void BmpToGrayGpu(const Matrix &A, Matrix &C)
{
	//kopiuje macierz Input do globalnej pamieci urzadzenia

	Matrix dA;
	dA.width=A.width;
	dA.height=A.height;
	size_t size=A.width*A.height*sizeof(int);
   	hipMalloc((void**)&dA.elementsRed, size);
	hipMemcpy(dA.elementsRed, A.elementsRed, size, hipMemcpyHostToDevice);
	hipMalloc((void**)&dA.elementsGreen, size);
	hipMemcpy(dA.elementsGreen, A.elementsGreen, size, hipMemcpyHostToDevice);
	hipMalloc((void**)&dA.elementsBlue, size);
	hipMemcpy(dA.elementsBlue, A.elementsBlue, size, hipMemcpyHostToDevice);

	//przydzielam macierz Output w globalnej pamieci urzadzenia

	Matrix dC;
	dC.width=C.width;
	dC.height=C.height;
	size=C.width*C.height*sizeof(int);
	hipMalloc((void**)&dC.elementsRed, size);
	hipMemcpy(dC.elementsRed, C.elementsRed, size, hipMemcpyHostToDevice);
	hipMalloc((void**)&dC.elementsGreen, size);
	hipMemcpy(dC.elementsGreen, C.elementsGreen, size, hipMemcpyHostToDevice);
	hipMalloc((void**)&dC.elementsBlue, size);
	hipMemcpy(dC.elementsBlue, C.elementsBlue, size, hipMemcpyHostToDevice);
   	
	//preparuje srodowisko i wywoluje kernel

	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(A.height / dimBlock.x + 1, A.width / dimBlock.y + 1);
	
	BmpToGrayKernel<<<dimGrid, dimBlock>>>(dA, dC);
	hipDeviceSynchronize();
	
	//odbieram obliczona macierz C z pamieci globalnej urzadzenia

	hipMemcpy(C.elementsRed, dC.elementsRed, size, hipMemcpyDeviceToHost);
	hipMemcpy(C.elementsGreen, dC.elementsGreen, size, hipMemcpyDeviceToHost);
	hipMemcpy(C.elementsBlue, dC.elementsBlue, size, hipMemcpyDeviceToHost);

	//zwalniam pamiec

	hipFree(dA.elementsRed);
	hipFree(dA.elementsGreen);
	hipFree(dA.elementsBlue);
	hipFree(dC.elementsRed);
	hipFree(dC.elementsGreen);
	hipFree(dC.elementsBlue);
}

// Kernel
__global__ void BmpToGrayKernel(Matrix A, Matrix C) {
	//kazdy watek oblicza jeden element macierzy C
	//akumulujac wynik w zmienej cvalue

	int j=blockIdx.y*blockDim.y+threadIdx.y; //row
	int i=blockIdx.x*blockDim.x+threadIdx.x; //col

	if (i==0 || j==0)
		return;

	if (i>=(A.height-1)||j>=(A.width-1))
		return;
	C.elementsRed[i*C.width+j]=0.2126*A.elementsRed[i*A.width+j] + 0.7152*A.elementsGreen[i*A.width+j]+0.0722*A.elementsBlue[i*A.width+j];
	C.elementsGreen[i*C.width+j]=0.2126*A.elementsRed[i*A.width+j] + 0.7152*A.elementsGreen[i*A.width+j]+0.0722*A.elementsBlue[i*A.width+j];
	C.elementsBlue[i*C.width+j]=0.2126*A.elementsRed[i*A.width+j] + 0.7152*A.elementsGreen[i*A.width+j]+0.0722*A.elementsBlue[i*A.width+j];
}

int main(int argc, char** argv)
{
	BMP Input, Output;
	double czas=0;
    	Input.ReadFromFile("in.bmp");
    	picWidth = Input.TellWidth();
	picHeight = Input.TellHeight();
	Output.SetSize( Input.TellWidth() , Input.TellHeight() );
	Output.SetBitDepth(24);

    int	 devCnt;
    hipGetDeviceCount(&devCnt);
    if(devCnt == 0) {
		perror("No CUDA devices available -- exiting.");
		return 1;
    }

	Matrix A, C;

	alokujMacierz(A);
	alokujMacierz(C);
	ConvertBmpToMatrix(Input, A);

	clock_t begin=clock(), end;

	BmpToGrayGpu(A, C);

	end=clock();
	czas=(double)(end-begin)/CLOCKS_PER_SEC;
	printf("Czas obliczen na GPU = %.9f sekund\n", czas);

	ConvertMatrixToBmp(Output, C);
	zwolnijMacierz(A);
	zwolnijMacierz(C);
	
	Output.WriteToFile("in2.bmp");
    return 0;
}
