#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <>
#include<time.h>

#define  N   		10000000
#define  BLOCK_SIZE	1024
float 	   hArray[N];
float     *dArray;
int 	   blocks;
float eps = 0.00000000001, pi_CPU, pi_GPU;

clock_t cpu_start, cpu_stop, gpu_start, gpu_stop, gpup_start, gpup_stop, gpue_start, gpue_stop, gpul_start, gpul_stop;
double cpu_ile, gpu_ile, gpup_ile, gpue_ile, gpul_ile;

void prologue(void) {
	hipMemset(hArray, 0, sizeof(hArray));
	for (int i = 0; i < N; i++) {
		hArray[i] = i + 1;
	}
	hipMalloc((void**)&dArray, sizeof(hArray));
	hipMemcpy(dArray, hArray, sizeof(hArray), hipMemcpyHostToDevice);
}

void epilogue(void) {
	hipMemcpy(hArray, dArray, sizeof(hArray), hipMemcpyDeviceToHost);
	hipFree(dArray);
}

// Kernel
__global__ void pow3(float *A) {
	int x = blockDim.x * blockIdx.x + threadIdx.x;

	if (x < N)
		A[x] = (4 * A[x] * A[x]) / ((4 * A[x] * A[x]) - 1.0);
}


void ilo_CPU_pi() {
	int flag = 1, n = 1;
	float wynik = 1.0, temp = 0.0, roznica = 0.0, wynik_pop = 0;

	do {
		wynik_pop = wynik;
		temp = (4.0*n*n) / ((4.0*n*n) - 1.0);
		wynik = wynik * temp;
		roznica = wynik - wynik_pop;

		if (abs(roznica) < eps) {
			flag = 0;
		}
		temp = 0;
		n++;
	} while (flag == 1);
	pi_CPU = wynik * 2.0;
}

void ilo_GPU_pi() {
	int flag = 1, n = 0;
	float wynik = 1.0, roznica = 0.0, wynik_pop = 0;

	do {
		wynik_pop = wynik;
		wynik = wynik * hArray[n];
		roznica = wynik - wynik_pop;

		if (abs(roznica) < eps) {
			flag = 0;
		}
		n++;
	} while (flag == 1);
	pi_GPU = wynik * 2.0;
}



int main(int argc, char** argv)
{
	int	 devCnt;

	hipGetDeviceCount(&devCnt);
	if (devCnt == 0) {
		perror("No CUDA devices available -- exiting.");
		return 1;
	}


	cpu_start = clock();

	for (int i = 0; i < N; i++) {
		hArray[i] = i + 1;
	}

	ilo_CPU_pi();

	cpu_stop = clock();


	cpu_ile = (double)1000 * (cpu_stop - cpu_start) / CLOCKS_PER_SEC;

	gpu_start = clock();

	gpu_start = clock();
	gpup_start = clock();
	prologue();
	gpup_stop = clock();

	gpul_start = clock();
	blocks = N / BLOCK_SIZE;
	if (N % BLOCK_SIZE) {
		blocks++;
	}
	pow3 << <blocks, BLOCK_SIZE >> > (dArray);
	hipDeviceSynchronize();
	gpul_stop = clock();
	gpue_start = clock();
	epilogue();
	gpue_stop = clock();


	ilo_GPU_pi();
	gpu_stop = clock();

	gpu_ile = (double)1000 * (gpu_stop - gpu_start) / CLOCKS_PER_SEC;
	gpup_ile = (double)1000 * (gpup_stop - gpup_start) / CLOCKS_PER_SEC;
	gpue_ile = (double)1000 * (gpue_stop - gpue_start) / CLOCKS_PER_SEC;
	gpul_ile = (double)1000 * (gpul_stop - gpul_start) / CLOCKS_PER_SEC;



	printf("  Lab 1. Zad 2. Iloczyn PI\n\n");

	printf("  pi GPU:    %f   \n", pi_GPU);
	printf("  pi CPU:    %f \n\n", pi_CPU);

	printf("  N:            %d \n", N);
	printf("  BLOCK SIZE:   %d \n", BLOCK_SIZE);
	printf("  Czas CPU:     %f [ms]\n", cpu_ile);
	printf("  Czas GPU:     %f [ms]\n", gpu_ile);
	printf("  GPU prolog:   %f [ms]\n", gpup_ile);
	printf("  GPU operacje: %f [ms]\n", gpul_ile);
	printf("  GPU epilog:   %f [ms]\n", gpue_ile);

	return 0;
}