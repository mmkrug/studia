#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <>
#include<time.h>
#include<stdlib.h>

#define N 2
#define  BLOCK_SIZE 64

typedef struct {
	int width;
	int height;
	int stride;
	float *elements;
} Matrix;



clock_t start_gpu_2, stop_gpu_2, start_gpu_1, stop_gpu_1;

double czas_gpu_2, czas_gpu_1;


// funkcja do odczytywania warto�ci elementu wskazanej macierzy

__device__ float GetElement(const Matrix A, int row, int col)
{
	return A.elements[row * A.stride + col];
}

// funkcja do zapisywania warto�ci elementu wskazanej macierzy
__device__ void SetElement(Matrix A, int row, int col, float value)
{
	A.elements[row * A.stride + col] = value;
}

// wykreowanie opisu podmacierzy o rozmiarze BLOCK_SIZExBLOCK_SIZE, kt�ra
// ulokowana jest col podmacierzy w prawo i row podmacierzy w d�
// licz�c od lewego wierzcho�ka danej macierzy
__device__ Matrix GetSubMatrix(Matrix A, int row, int col)
{
	Matrix Asub;
	Asub.width = BLOCK_SIZE;
	Asub.height = BLOCK_SIZE;
	Asub.stride = A.stride;
	Asub.elements = &A.elements[A.stride * BLOCK_SIZE * row + BLOCK_SIZE * col];
	return Asub;
}

__global__ void MatMulKernelWspol(Matrix A, Matrix B, Matrix C) {

	// ustalenie numeru wiersza i kolumny wewn�trz bloku
	int blockRow = blockIdx.y;
	int blockCol = blockIdx.x;

	float Cvalue = 0;

	// ka�dy blok oblicza jedn� podmacierz Csub macierzy C
	Matrix Csub = GetSubMatrix(C, blockRow, blockCol);

	// ustalenie numeru wiersza i kolumny wewn�trz w�tku
	int row = threadIdx.y;
	int col = threadIdx.x;

	// iterujemy wszystkie podmacierze A i B, kt�re
	// s� potrzebne do obliczenia Csub � mno�ymy ze sob� ka�d� par�
	// podmacierzy i akumulujemy wynik
	for (int m = 0; m < (A.width / BLOCK_SIZE); ++m) {

		// kreujemy podmacierz Asub macierzy A
		Matrix Asub = GetSubMatrix(A, blockRow, m);

		// kreujemy podmacierz Bsub macierzy B
		Matrix Bsub = GetSubMatrix(B, m, blockCol);

		// deklarujemy obszar pami�ci dzielonej dla podmacierzy Asub i Bsub
		__shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
		__shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

		// za�aduj Asub i Bsub z pami�ci globalnej do dzielonej
		// (ka�dy w�tek �aduje jeden element z ka�dej podmacierzy)
		As[row][col] = GetElement(Asub, row, col);
		Bs[row][col] = GetElement(Bsub, row, col);

		// poczekajmy, a� wszystkie dane zostan� skopiowane
		__syncthreads();

		// mno�ymy Asub i Bsub
		for (int e = 0; e < BLOCK_SIZE; ++e)
			Cvalue += As[row][e] * Bs[e][col];

		// poczekajmy, a� obliczenia zostan� zako�czone zanim zabierzemy
		// si� za przetwarzanie nast�pnej podmacierzy
		__syncthreads();
	}

	// odsy�amy obliczone Cvalue do pami�ci urz�dzenia
	SetElement(Csub, row, col, Cvalue);
}

__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C) {

	// ka�dy w�tek oblicza jeden element macierzy C
	// akumuluj�c wynik w zmiennej Cvalue
	float Cvalue = 0;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	for (int e = 0; e < A.width; ++e)
		Cvalue += A.elements[row * A.width + e] * B.elements[e * B.width + col];
	C.elements[row * C.width + col] = Cvalue;
}

void MatMul(const Matrix A, const Matrix B, Matrix C) {
	// kopiujemy macierze A i B to globalnej pami�ci urz�dzenia
	// najpierw A

	Matrix d_A;
	d_A.width = A.width;

	d_A.height = A.height;
	size_t size = A.width * A.height * sizeof(float);
	hipMalloc((void **)&d_A.elements, size);
	hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);

	// potem B
	Matrix d_B;
	d_B.width = B.width;
	d_B.height = B.height;
	size = B.width * B.height * sizeof(float);
	hipMalloc((void **)&d_B.elements, size);
	hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);

	// przydzielamy macierz C w globalnej pami�ci urz�dzenia
	Matrix d_C;
	d_C.width = C.width;
	d_C.height = C.height;
	size = C.width * C.height * sizeof(float);
	hipMalloc((void**)&d_C.elements, size);

	// preparujemy �rodowisko i wywo�ujemy kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);

	start_gpu_2 = clock();
	MatMulKernel << <dimGrid, dimBlock >> > (d_A, d_B, d_C);
	hipDeviceSynchronize();
	stop_gpu_2 = clock();
	czas_gpu_2 = (double)1000 * (stop_gpu_2 - start_gpu_2) / CLOCKS_PER_SEC;

	// odbieramy obliczon� macierz C z pami�ci globalnej urz�dzenia
	hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);

	printf("  Czas GPU: %f[ms]\n", czas_gpu_2);

	// zwalniamy pami�� 
	hipFree(d_A.elements);
	hipFree(d_B.elements);
	hipFree(d_C.elements);
}


void MatMulWspol(const Matrix A, const Matrix B, Matrix C) {
	// kopiujemy macierze A i B to globalnej pami�ci urz�dzenia
	// najpierw A

	Matrix d_A;
	d_A.width = d_A.stride = A.width;
	d_A.height = A.height;
	size_t size = A.width * A.height * sizeof(float);
	hipMalloc((void **)&d_A.elements, size);
	hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);

	// potem B
	Matrix d_B;
	d_B.width = d_B.stride = B.width;
	d_B.height = B.height;
	size = B.width * B.height * sizeof(float);
	hipMalloc((void **)&d_B.elements, size);
	hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);

	// przydzielamy macierz C w globalnej pami�ci urz�dzenia
	Matrix d_C;
	d_C.width = d_C.stride = C.width;
	d_C.height = C.height;
	size = C.width * C.height * sizeof(float);
	hipMalloc((void**)&d_C.elements, size);

	// preparujemy �rodowisko i wywo�ujemy kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);

	start_gpu_1 = clock();
	MatMulKernelWspol << <dimGrid, dimBlock >> > (d_A, d_B, d_C);
	hipDeviceSynchronize();
	stop_gpu_1 = clock();
	czas_gpu_1 = (double)1000 * (stop_gpu_1 - start_gpu_1) / CLOCKS_PER_SEC;

	// odbieramy obliczon� macierz C z pami�ci globalnej urz�dzenia
	hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);

	printf("  Czas GPU z pam. dzielona: %f[ms]\n", czas_gpu_1);


	// zwalniamy pami�� 
	hipFree(d_A.elements);
	hipFree(d_B.elements);
	hipFree(d_C.elements);
}

int main(int argc, char** argv)
{
	printf("  Lab 2. Zad 2. Macierze\n\n");
	printf("  N:            %d \n", N);
	printf("  BLOCK SIZE:   %d \n", BLOCK_SIZE);
	int  devCnt;

	Matrix A, B, C;
	hipGetDeviceCount(&devCnt);
	if (devCnt == 0) {
		perror("No CUDA devices available -- exiting.");
		return 1;
	}
	A.width = N;
	A.height = N;
	A.elements = (float*)malloc(A.width*A.height * sizeof(float));
	for (int i = 0; i < A.width*A.height; i++) {
		A.elements[i] = ((float)(rand() % 100) / 100) + (rand() % 50);
	}
	B.width = N;
	B.height = N;
	B.elements = (float*)malloc(B.width*B.height * sizeof(float));
	for (int i = 0; i < B.width*B.height; i++) {
		B.elements[i] = ((float)(rand() % 100) / 100) + (rand() % 50);
	}
	C.width = B.width;
	C.height = A.height;
	C.elements = (float*)malloc(C.width*C.height * sizeof(float));

	MatMulWspol(A, B, C);
	MatMul(A, B, C);

	free(A.elements);
	free(B.elements);
	free(C.elements);
}