#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <>
#include<time.h>
#include<stdlib.h>


#define N 128
#define  BLOCK_SIZE	8

typedef struct {
	int width;
	int height;
	float *elements;
} Matrix;


clock_t start_cpu, stop_cpu, start_gpu, stop_gpu;

double czas_cpu, czas_gpu;


__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C) {

	// ka�dy w�tek oblicza jeden element macierzy C
	// akumuluj�c wynik w zmiennej Cvalue
	float Cvalue = 0;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	for (int e = 0; e < A.width; ++e)
		Cvalue += A.elements[row * A.width + e] * B.elements[e * B.width + col];
	C.elements[row * C.width + col] = Cvalue;
}

void MatMul(const Matrix A, const Matrix B, Matrix C) {
	// kopiujemy macierze A i B to globalnej pami�ci urz�dzenia
	// najpierw A

	Matrix d_A;
	d_A.width = A.width;

	d_A.height = A.height;
	size_t size = A.width * A.height * sizeof(float);
	hipMalloc((void **)&d_A.elements, size);
	hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);

	// potem B 
	Matrix d_B;
	d_B.width = B.width;
	d_B.height = B.height;
	size = B.width * B.height * sizeof(float);
	hipMalloc((void **)&d_B.elements, size);
	hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);

	// przydzielamy macierz C w globalnej pami�ci urz�dzenia
	Matrix d_C;
	d_C.width = C.width;
	d_C.height = C.height;
	size = C.width * C.height * sizeof(float);
	hipMalloc((void**)&d_C.elements, size);

	// preparujemy �rodowisko i wywo�ujemy kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);

	start_gpu = clock();

	MatMulKernel << <dimGrid, dimBlock >> > (d_A, d_B, d_C);
	hipDeviceSynchronize();

	stop_gpu = clock();

	czas_gpu = (double)1000 * (stop_gpu - start_gpu) / CLOCKS_PER_SEC;

	// odbieramy obliczon� macierz C z pami�ci globalnej urz�dzenia
	hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);

	printf("  Czas GPU: %f[ms]\n", czas_gpu);

	printf("\n");
	// zwalniamy pami�� 
	hipFree(d_A.elements);
	hipFree(d_B.elements);
	hipFree(d_C.elements);
}

int main(int argc, char** argv)
{
	printf("  Lab 2. Zad 1. Macierze\n\n");
	printf("  N:            %d \n", N);
	printf("  BLOCK SIZE:   %d \n", BLOCK_SIZE);
	int	 devCnt;

	Matrix A, B, C;

	hipGetDeviceCount(&devCnt);
	if (devCnt == 0) {
		perror("No CUDA devices available -- exiting.");
		return 1;
	}

	A.width = N;
	A.height = N;
	A.elements = (float*)malloc(A.width*A.height * sizeof(float));
	for (int i = 0; i < A.width*A.height; i++) {
		A.elements[i] = ((float)(rand() % 100) / 100) + (rand() % 50);
	}

	B.width = N;
	B.height = N;
	B.elements = (float*)malloc(B.width*B.height * sizeof(float));
	for (int i = 0; i < B.width*B.height; i++) {
		B.elements[i] = ((float)(rand() % 100) / 100) + (rand() % 50);
	}

	C.width = B.width;
	C.height = A.height;
	C.elements = (float*)malloc(C.width*C.height * sizeof(float));


	float **A2D = new float*[A.width];
	for (int i = 0; i < A.height; i++)
		A2D[i] = new float[A.width];

	float **B2D = new float*[A.width];
	for (int i = 0; i < A.height; i++)
		B2D[i] = new float[A.width];

	float **C2D = new float*[A.width];
	for (int i = 0; i < A.height; i++)
		C2D[i] = new float[A.width];




	for (int i = 0; i < A.width; i++) {
		for (int j = 0; j < A.height; j++) {
			A2D[i][j] = A.elements[i*A.width + j];
		}
	}
	for (int i = 0; i < B.width; i++) {
		for (int j = 0; j < B.height; j++) {
			B2D[i][j] = B.elements[i*B.width + j];
		}
	}
	for (int i = 0; i < C.width; i++) {
		for (int j = 0; j < C.height; j++) {
			C2D[i][j] = 0;
		}
	}

	start_cpu = clock();

	for (int i = 0; i < A.height; i++) {
		for (int j = 0; j < A.width; j++) {
			for (int k = 0; k < B.width; k++) {
				C2D[i][k] += A2D[i][j] * B2D[j][k];
			}
		}
	}

	stop_cpu = clock();

	czas_cpu = (double)1000 * (stop_cpu - start_cpu) / CLOCKS_PER_SEC;
	printf("  Czas CPU %.2f[ms]\n", czas_cpu);



	MatMul(A, B, C);



	for (int i = 0; i < A.width; i++) {
		delete[] A2D[i];
		delete[] B2D[i];
		delete[] C2D[i];
	}

	delete[] A2D;
	delete[] B2D;
	delete[] C2D;

	free(A.elements);
	free(B.elements);
	free(C.elements);
}