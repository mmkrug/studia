#include "hip/hip_runtime.h"
#include "EasyBMP.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

using namespace std;

void bmpToGray(BMP Input, BMP &Output, int picWidth, int picHeight)
{
	for (int i = 0; i < picWidth; ++i)
		for (int j = 0; j < picHeight; ++j) {

			Output(i,j)->Red = 0.2126*Input(i,j)->Red + 0.7152*Input(i,j)->Green + 0.0722*Input(i, j)->Blue;
			Output(i,j)->Blue = 0.2126*Input(i,j)->Red + 0.7152*Input(i,j)->Green + 0.0722*Input(i, j)->Blue;
			Output(i,j)->Green = 0.2126*Input(i,j)->Red + 0.7152*Input(i,j)->Green + 0.0722*Input(i,j)->Blue;
	}
}

int main( int argc, char* argv[] )
{
double czas=0;
BMP Input;
Input.ReadFromFile("in.bmp");
BMP Output;
int picWidth = Input.TellWidth();
int picHeight = Input.TellHeight();
Output.SetSize( Input.TellWidth() , Input.TellHeight() );
Output.SetBitDepth(24);

clock_t begin=clock(), end;
bmpToGray(Input, Output, picWidth, picHeight);
end=clock();
czas=(double)(end-begin)/CLOCKS_PER_SEC;
printf("Czas obliczen na CPU = %.9f sekund\n", czas);
Output.WriteToFile("in2.bmp");
return 0;
}
